#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include "time.h"
#include "matmul.cuh"

using namespace std;

int main(int argc, char *argv[]) {
    srand((unsigned int)time(NULL));
    long n = atol(argv[1]);
    long threads_per_block = atol(argv[2]);
    long totalSize = n * n * sizeof(float);

    float *dA, *dB, *dC;
    // hipMallocManaged() simplifies memory access by eliminating the need for explicit memory allocations on host and device
    hipMallocManaged((void **)&dA, totalSize);
    hipMallocManaged((void **)&dB, totalSize);
    hipMallocManaged((void **)&dC, totalSize);
    for (int i = 0; i < n * n; i++) {
        dA[i] = ((float)rand() / (RAND_MAX)) * 2 - 1;  // [-1.0, 1.0]
        dB[i] = ((float)rand() / (RAND_MAX)) * 2 - 1;
        dC[i] = 0.0;
    }

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matmul(dA, dB, dC, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    printf("%f\n", dC[n * n - 1]);
    printf("%f\n", ms);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}
