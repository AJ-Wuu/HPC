#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include "time.h"
#include "stencil.cuh"

using namespace std;

int main(int argc, char *argv[]) {
    srand((unsigned int)time(NULL));
    long n = atol(argv[1]);
    long R = atol(argv[2]);
    long threads_per_block = atol(argv[3]);

    float *image, *mask, *output;
    hipMallocManaged((void **)&image, n * sizeof(float));
    hipMallocManaged((void **)&mask, (2 * R + 1) * sizeof(float));
    hipMallocManaged((void **)&output, n * sizeof(float));
    for (int i = 0; i < n; i++) {
        image[i] = ((float)rand() / (RAND_MAX)) * 2 - 1;  // [-1.0, 1.0]
    }
    for (int i = 0; i < 2 * R + 1; i++) {
        mask[i] = ((float)rand() / (RAND_MAX)) * 2 - 1;
    }

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    stencil(image, mask, output, n, R, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    printf("%f\n", output[n - 1]);
    printf("%f\n", ms);

    hipFree(image);
    hipFree(mask);
    hipFree(output);
}