#include "scan.cuh"
#include "time.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

using namespace std;

int main(int argc, char *argv[]) {
    srand((unsigned int)time(NULL));
    long n = atol(argv[1]);
    long threads_per_block = atol(argv[2]);

    float *input, *output;
    hipMallocManaged(&input, n * sizeof(float));
    hipMallocManaged(&output, n * sizeof(float));
    for (int i = 0; i < n; i++) {
        input[i] = ((float)rand() / (RAND_MAX)) * 2 - 1;  // [-1.0, 1.0]
    }

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    scan(input, output, n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

    printf("%f\n", output[n - 1]);
    printf("%f\n", ms);

    hipFree(input);
    hipFree(output);
}