#include "mmul.h"
#include "time.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

using namespace std;

int main(int argc, char *argv[]) {
    srand((unsigned int)time(NULL));
    long n = atol(argv[1]);
    long n_tests = atol(argv[2]);
    long totalSize = n * n * sizeof(float);

    float *A, *B, *C; // stored in managed memory
    hipMallocManaged(&A, totalSize);
    hipMallocManaged(&B, totalSize);
    hipMallocManaged(&C, totalSize);
    for (int j = 0; j < n; j++) {
        for (int i = 0; i < n; i++) {
            int index = i * n + j; // column-major order 
            A[index] = ((float)rand() / (RAND_MAX)) * 2 - 1;  // [-1.0, 1.0]
            B[index] = ((float)rand() / (RAND_MAX)) * 2 - 1;
            C[index] = 0.0;
        }
    }

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float ms_total = 0.0, ms;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    for (int i = 0; i < n_tests; i++) {
        hipEventRecord(start, 0);
        mmul(handle, A, B, C, n);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        ms_total += ms;

        /*
        for (int j = 0; j < n * n; j++) { // re-initialize C -- takes too long to finish
            C[j] = 0.0;
        }
        */
    }

    printf("%f\n", ms_total / n_tests);

    hipFree(A);
    hipFree(B);
    hipFree(C);
    hipblasDestroy(handle);
}