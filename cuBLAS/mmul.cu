#include "mmul.h"
#include <hipblas.h>

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n) {
    /*
    hipblasStatus_t hipblasSgemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const float *alpha,
                           const float *A, int lda,
                           const float *B, int ldb,
                           const float *beta,
                           float *C, int ldc)
    */
    int lda = n, ldb = n, ldc = n;
    const float alpha = 1.0, beta = 1.0;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, lda, B, ldb, &beta, C, ldc);
    hipDeviceSynchronize();
}