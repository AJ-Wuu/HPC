
#include <hip/hip_runtime.h>
__global__ void vscale(const float *a, float *b, unsigned int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {  // validate the index
        b[index] *= a[index];
    }
}