#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>

#include "time.h"

using namespace std;

const int ARRAY_SIZE = 16;
const int nBlocks = 2;
const int nThreads = 8;

__global__ void task2(int *dA, int a) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int x = threadIdx.x;
    int y = blockIdx.x;
    dA[index] = a * x + y;
}

int main() {
    srand((unsigned int)time(NULL));
    int a = (int)rand();
    int totalSize = ARRAY_SIZE * sizeof(int);

    int *dA;
    hipMalloc((void **)&dA, totalSize);

    task2<<<nBlocks, nThreads>>>(dA, a);
    hipDeviceSynchronize();

    int *hA;
    hA = (int *)malloc(totalSize);
    hipMemcpy(hA, dA, totalSize, hipMemcpyDeviceToHost);

    for (int i = 0; i < ARRAY_SIZE; i++) {
        if (i == ARRAY_SIZE - 1) {
            std::printf("%d", hA[i]);
        } else {
            std::printf("%d ", hA[i]);
        }
    }

    hipFree(dA);
    free(hA);
}
